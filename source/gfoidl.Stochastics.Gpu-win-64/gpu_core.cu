#include "gpu_core.h"
#include <hip/hip_runtime.h>
//-----------------------------------------------------------------------------
const bool gpu_available()
{
    int deviceCount;
    hipError_t errorId = hipGetDeviceCount(&deviceCount);

    return errorId == hipSuccess
        && deviceCount > 0;
}
//-----------------------------------------------------------------------------
const char* gpu_get_error_string(const int errorCode)
{
    return hipGetErrorString(static_cast<hipError_t>(errorCode));
}
//-----------------------------------------------------------------------------
const int gpu_sample_calc_stats(double* sample, const int sampleSize, SampleStats* sampleStats)
{
    sampleStats->Delta = 1;
    sampleStats->Kurtosis = 2;
    sampleStats->Max = 3;
    sampleStats->Mean = 4;
    sampleStats->Min = 5;
    sampleStats->Skewness = 6;
    sampleStats->VarianceCore = 7;

    return 0;
}
