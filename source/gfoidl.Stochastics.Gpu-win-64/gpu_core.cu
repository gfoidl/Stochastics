#include "gpu_core.h"
#include <hip/hip_runtime.h>
#include "kernel.h"

#if defined(DEBUG) || defined(_DEBUG)
    #include <stdio.h>
    #include <assert.h>
#endif
//-----------------------------------------------------------------------------
// Forward declarations
inline hipError_t checkCuda(hipError_t result);
//-----------------------------------------------------------------------------
const bool gpu_available()
{
    int deviceCount;
    hipError_t errorId = hipGetDeviceCount(&deviceCount);

    return errorId == hipSuccess
        && deviceCount > 0;
}
//-----------------------------------------------------------------------------
const char* gpu_get_error_string(const int errorCode)
{
    return hipGetErrorString(static_cast<hipError_t>(errorCode));
}
//-----------------------------------------------------------------------------
const int gpu_sample_calc_stats(double* sample, const int sampleSize, SampleStats* sampleStats)
{
    double*      deviceSample;
    SampleStats* deviceSampleStats;

    try
    {
        checkCuda(hipMalloc(&deviceSample, sizeof(double) * sampleSize));
        checkCuda(hipMalloc(&deviceSampleStats, sizeof(SampleStats)));

        checkCuda(hipMemcpy(deviceSample, sample, sizeof(double) * sampleSize, hipMemcpyHostToDevice));
        checkCuda(hipMemset(deviceSampleStats, 0, sizeof(SampleStats)));

        const int blockSize = 256;
        int numBlocks       = (sampleSize + blockSize - 1) / blockSize;

#if defined(DEBUG) || defined(_DEBUG)
        printf("blockSize: %d\nnumBlocks: %d\n", blockSize, numBlocks);
#endif

        // For final fixup of values a separate kernel is queued to the device.
        // Otherwise there's no way of syncing all threads in the grid.
        Kernel::CalculateAverageAndVarianceCore<<<numBlocks, blockSize>>>(deviceSample, sampleSize, deviceSampleStats);
        Kernel::CalculateAverageAndVarianceCoreFinal<<<1, 1>>>(deviceSampleStats, sampleSize);

        //checkCuda(hipDeviceSynchronize());       // not necessary
        checkCuda(hipMemcpy(sampleStats, deviceSampleStats, sizeof(SampleStats), hipMemcpyDeviceToHost));

        checkCuda(hipFree(deviceSample));
        checkCuda(hipFree(deviceSampleStats));
    }
    catch (const int e)
    {
        return e;
    }

    return 0;
}
//-----------------------------------------------------------------------------
hipError_t checkCuda(hipError_t result)
{
    if (result != hipSuccess)
        throw static_cast<int>(result);

    return result;
}
