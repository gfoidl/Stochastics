#include "hip/hip_runtime.h"
#include "kernel.h"
#include "kernel_utils.h"
//-----------------------------------------------------------------------------
namespace Kernel
{
    __global__
    void CalculateAverageAndVarianceCore(const double* sample, const int n, SampleStats* sampleStats)
    {
        const int index  = blockDim.x * blockIdx.x + threadIdx.x;
        const int stride = gridDim.x * blockDim.x;

        double avg      = 0;
        double variance = 0;

        for (int i = index; i < n; i += stride)
        {
            avg      += sample[i];
            variance += sample[i] * sample[i];
        }

        Utils::TwoDoubles twoDoubles {avg, variance};
        twoDoubles = Utils::BlockReduceSum(twoDoubles);

        // Final sum in first thread of each block
        if (threadIdx.x == 0)
        {
            atomicAdd(&sampleStats->Mean        , twoDoubles.A);
            atomicAdd(&sampleStats->VarianceCore, twoDoubles.B);
        }
    }
    //-----------------------------------------------------------------------------
    __global__
    void CalculateAverageAndVarianceCoreFinal(SampleStats* sampleStats, const int n)
    {
        const int index = blockDim.x * blockIdx.x + threadIdx.x;

        if (index == 0)
        {
            double avg                 = sampleStats->Mean / n;
            sampleStats->Mean          = avg;
            sampleStats->VarianceCore -= n * avg*avg;
        }
    }
    //-----------------------------------------------------------------------------
    __global__
    void CalculateDeltaSkewnessKurtosis(const double* sample, const int n, SampleStats* sampleStats)
    {
        const int index  = blockDim.x * blockIdx.x + threadIdx.x;
        const int stride = gridDim.x * blockDim.x;

        double avg      = sampleStats->Mean;
        double delta    = 0;
        double skewness = 0;
        double kurtosis = 0;

        for (int i = index; i < n; i += stride)
        {
            double t = sample[i] - avg;
            double t1 = t * t*t;

            delta    += abs(t);
            skewness += t1;
            kurtosis += t1 * t;
        }

        Utils::ThreeDoubles threeDoubles {delta, skewness,kurtosis};
        threeDoubles = Utils::BlockReduceSum(threeDoubles);

        // Final sum in first thread of each block
        if (threadIdx.x == 0)
        {
            atomicAdd(&sampleStats->Delta, threeDoubles.A);
            atomicAdd(&sampleStats->Skewness, threeDoubles.B);
            atomicAdd(&sampleStats->Kurtosis, threeDoubles.C);
        }
    }
}
